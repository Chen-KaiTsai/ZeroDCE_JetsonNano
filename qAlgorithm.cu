#include "main.cuh"

void DCE::initMem() {
    INDATA = (RGBIOData*)malloc(sizeof(RGBIOData_t));
    
    OUTDATA = (RGBIOData*)malloc(sizeof(RGBIOData_t));

    NETIO = (qNetIO_t*)malloc(sizeof(qNetIO_t));
#ifdef CPU_UPSAMPLE
    UPSBUFFER = (qEnhancedParam_t*)malloc(sizeof(qEnhancedParam_t));
    PARAM = (qEnhancedParam_t*)malloc(sizeof(qEnhancedParam_t));
#endif
    CONVW01 = (qWConv1st_t*)malloc(sizeof(qWConv1st_t));
    CONVB01 = (qBConv1st_t*)malloc(sizeof(qBConv1st_t));
    CONVW02 = (qWConv2nd_t*)malloc(sizeof(qWConv2nd_t));
    CONVB02 = (qBConv2nd_t*)malloc(sizeof(qBConv2nd_t));
    CONVW03 = (qWConv3rd_t*)malloc(sizeof(qWConv3rd_t));
    CONVB03 = (qBConv3rd_t*)malloc(sizeof(qBConv3rd_t));
}

void DCE::cleanMem() {
    // Free CPU Memory
    if (INDATA != nullptr) {
        free(INDATA);
        INDATA = nullptr;
    }
    if (OUTDATA != nullptr) {
        free(OUTDATA);
        OUTDATA = nullptr;
    }
}

void DCE::qNormNDownSample() {
    hipError_t error;

    error = hipMalloc(&dINDATA, sizeof(RGBIOData_t));
    if (error != hipSuccess) {
        printf("Error dINDATA hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dINDATA, INDATA, sizeof(RGBIOData_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dINDATA hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dNORM, sizeof(qNormImg_t));
    if (error != hipSuccess) {
        printf("Error dNORM hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Run dNorm kernel
    //constexpr unsigned int jobSize = IMG_HIGHT * IMG_WIDTH * IMG_CHANNEL;

    dim3 dimBlock {24, 40, 1};
    dim3 dimGrid;

    dimGrid.x = (IMG_HIGHT + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (IMG_WIDTH + dimBlock.y - 1) / dimBlock.y;
    dimGrid.z = (IMG_CHANNEL + dimBlock.z - 1) / dimBlock.z;

    //printf("dimBlock {%d, %d, %d} dimGrid {%d, %d, %d}\n\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

    dNorm<<<dimGrid, dimBlock>>>(dINDATA, dNORM);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dNorm %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

    if (dINDATA != nullptr) {
        error = hipFree(dINDATA);
        if (error != hipSuccess) {
            printf("Error dINDATA hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dINDATA = nullptr;
    }

    error = hipMalloc(&dNETIO, sizeof(qNetIO_t));
    if (error != hipSuccess) {
        printf("Error dNETIO hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    // Run dDownSample kernel
    dimBlock = {30, 32, 1};
    dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};

    dDownSample<<<dimGrid, dimBlock>>>(dNORM, dNETIO);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dDownSample %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void DCE::qConv1st() {
    hipError_t error;

    error = hipMalloc(&dCONVW01, sizeof(qWConv1st_t));
    if (error != hipSuccess) {
        printf("Error dCONVW01 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dCONVW01, CONVW01, sizeof(qWConv1st_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dCONVW01 hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dCONVB01, sizeof(qBConv1st_t));
    if (error != hipSuccess) {
        printf("Error dCONVB01 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dCONVB01, CONVB01, sizeof(qBConv1st_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dCONVB01 hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dFEATURE1, sizeof(qNetFeature_t));
    if (error != hipSuccess) {
        printf("Error dFEATURE1 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Run dConv1st kernel
    dim3 dimBlock = {18, 40, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};

    dConv1st<<<dimGrid, dimBlock>>>(dNETIO, dCONVW01, dCONVB01, dFEATURE1);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dConv1st %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

    // Free CONVW01, CONVB01
    if (CONVW01 != nullptr) {
        free(CONVW01);
        CONVW01 = nullptr;
    }
    if (CONVB01 != nullptr) {
        free(CONVB01);
        CONVB01 = nullptr;
    }

    // Free dCONVW01, dCONVB01
    if (dCONVW01 != nullptr) {
        error = hipFree(dCONVW01);
        if (error != hipSuccess) {
            printf("Error dCONVW01 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dCONVW01 = nullptr;
    }
    if (dCONVB01 != nullptr) {
        error = hipFree(dCONVB01);
        if (error != hipSuccess) {
            printf("Error dCONVB01 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dCONVB01 = nullptr;
    }
}

void DCE::qConv2nd() {
    hipError_t error;

    error = hipMalloc(&dCONVW02, sizeof(qWConv2nd_t));
    if (error != hipSuccess) {
        printf("Error dCONVW02 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dCONVW02, CONVW02, sizeof(qWConv2nd_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dCONVW02 hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dCONVB02, sizeof(qBConv2nd_t));
    if (error != hipSuccess) {
        printf("Error dCONVB02 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dCONVB02, CONVB02, sizeof(qBConv2nd_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dCONVB02 hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dFEATURE2, sizeof(qNetFeature_t));
    if (error != hipSuccess) {
        printf("Error dFEATURE2 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Run dConv1st kernel
    dim3 dimBlock = {18, 40, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};

    dConv2nd<<<dimGrid, dimBlock>>>(dFEATURE1, dCONVW02, dCONVB02, dFEATURE2);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dConv2nd %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

    // Free CONVW01, CONVB01
    if (CONVW02 != nullptr) {
        free(CONVW02);
        CONVW02 = nullptr;
    }
    if (CONVB02 != nullptr) {
        free(CONVB02);
        CONVB02 = nullptr;
    }

    // Free dCONVW01, dCONVB01
    if (dCONVW02 != nullptr) {
        error = hipFree(dCONVW02);
        if (error != hipSuccess) {
            printf("Error dCONVW02 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dCONVW02 = nullptr;
    }
    if (dCONVB02 != nullptr) {
        error = hipFree(dCONVB02);
        if (error != hipSuccess) {
            printf("Error dCONVB02 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dCONVB02 = nullptr;
    }
}

void DCE::qConv3rd() {
    hipError_t error;

    error = hipMalloc(&dCONVW03, sizeof(qWConv3rd_t));
    if (error != hipSuccess) {
        printf("Error dCONVW03 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dCONVW03, CONVW03, sizeof(qWConv3rd_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dCONVW03 hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    error = hipMalloc(&dCONVB03, sizeof(qBConv3rd_t));
    if (error != hipSuccess) {
        printf("Error dCONVB03 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dCONVB03, CONVB03, sizeof(qBConv3rd_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dCONVB03 hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Run dConv3rd
    dim3 dimBlock = {18, 40, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};
    dConv3rd<<<dimGrid, dimBlock>>>(dFEATURE1, dFEATURE2, dCONVW03, dCONVB03, dNETIO);

    // Release dFEATURE1 dFEATURE2 dCONVW3 dCONVB3
    if (CONVW03 != nullptr) {
        free(CONVW03);
        CONVW03 = nullptr;
    }
    if (CONVB03 != nullptr) {
        free(CONVB03);
        CONVB03 = nullptr;
    }

    if (dCONVW03 != nullptr) {
        error = hipFree(dCONVW03);
        if (error != hipSuccess) {
            printf("Error dCONVW03 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dCONVW03 = nullptr;
    }
    if (dCONVB03 != nullptr) {
        error = hipFree(dCONVB03);
        if (error != hipSuccess) {
            printf("Error dCONVB03 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dCONVB03 = nullptr;
    }
    if (dFEATURE1 != nullptr) {
        error = hipFree(dFEATURE1);
        if (error != hipSuccess) {
            printf("Error dFEATURE1 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dFEATURE1 = nullptr;
    }
    if (dFEATURE2 != nullptr) {
        error = hipFree(dFEATURE2);
        if (error != hipSuccess) {
            printf("Error dFEATURE2 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dFEATURE2 = nullptr;
    }

    // Copy NETIO to CPU
#ifdef CPU_UPSAMPLE
    error = hipMemcpy(NETIO, dNETIO, sizeof(qNetIO_t), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Error dNETIO hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Release NETIO
    if (dNETIO != nullptr) {
        error = hipFree(dNETIO);
        if (error != hipSuccess) {
            printf("Error dNETIO hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dNETIO = nullptr;
    }
#endif
}

#ifdef CPU_UPSAMPLE
void DCE::qUpSample()
{
    int coef[12] = {42, 128, 213, 298, 384, 469, 554, 640, 725, 810, 896, 981};
	for(int h = 0; h < DCE_HEIGHT; ++h)
	{
		int wi = 0;
		for(int d = 0; d < DSRATE / 2; ++d, ++wi)
		{
			UPSBUFFER->data[h][wi][0] = NETIO->data[h][0][0];
			UPSBUFFER->data[h][wi][1] = NETIO->data[h][0][1];
			UPSBUFFER->data[h][wi][2] = NETIO->data[h][0][2];
		}
		for(int w = 1; w < DCE_WIDTH; ++w)
		{
			for(int d = 0; d < DSRATE; ++d, ++wi)
			{
				UPSBUFFER->data[h][wi][0] = (coef[d] * (NETIO->data[h][w][0] - NETIO->data[h][w - 1][0]) >> 10) + NETIO->data[h][w - 1][0];
				UPSBUFFER->data[h][wi][1] = (coef[d] * (NETIO->data[h][w][1] - NETIO->data[h][w - 1][1]) >> 10) + NETIO->data[h][w - 1][1];
				UPSBUFFER->data[h][wi][2] = (coef[d] * (NETIO->data[h][w][2] - NETIO->data[h][w - 1][2]) >> 10) + NETIO->data[h][w - 1][2];
			}
		}
		for(int d = 0; d < DSRATE / 2; ++d, ++wi)
		{
			UPSBUFFER->data[h][wi][0] = NETIO->data[h][DCE_WIDTH - 1][0];
			UPSBUFFER->data[h][wi][1] = NETIO->data[h][DCE_WIDTH - 1][1];
			UPSBUFFER->data[h][wi][2] = NETIO->data[h][DCE_WIDTH - 1][2];
		}
	}

	int hi = 0;
	for(int d = 0; d < DSRATE / 2; ++d, ++hi)
	{
		for(int w = 0; w < IMG_WIDTH; ++w)
		{
			PARAM->data[hi][w][0] = UPSBUFFER->data[0][w][0];
			PARAM->data[hi][w][1] = UPSBUFFER->data[0][w][1];
			PARAM->data[hi][w][2] = UPSBUFFER->data[0][w][2];
		}
	}
	for(int h = 1; h < DCE_HEIGHT; ++h)
	{
		for(int d = 0; d < DSRATE; ++d, ++hi)
		{
			for(int w = 0; w < IMG_WIDTH; ++w)
			{
				PARAM->data[hi][w][0] = (coef[d] * (UPSBUFFER->data[h][w][0] - UPSBUFFER->data[h - 1][w][0]) >> 10) + UPSBUFFER->data[h - 1][w][0];
				PARAM->data[hi][w][1] = (coef[d] * (UPSBUFFER->data[h][w][1] - UPSBUFFER->data[h - 1][w][1]) >> 10) + UPSBUFFER->data[h - 1][w][1];
				PARAM->data[hi][w][2] = (coef[d] * (UPSBUFFER->data[h][w][2] - UPSBUFFER->data[h - 1][w][2]) >> 10) + UPSBUFFER->data[h - 1][w][2];
			}

		}
	}
	for(int d = 0; d < DSRATE / 2; ++d, ++hi)
	{
		for(int w = 0; w < IMG_WIDTH; ++w)
		{
			PARAM->data[hi][w][0] = UPSBUFFER->data[DCE_HEIGHT - 1][w][0];
			PARAM->data[hi][w][1] = UPSBUFFER->data[DCE_HEIGHT - 1][w][1];
			PARAM->data[hi][w][2] = UPSBUFFER->data[DCE_HEIGHT - 1][w][2];
		}
	}

    if (UPSBUFFER != nullptr) {
        free(UPSBUFFER);
        UPSBUFFER = nullptr;
    }
    if (NETIO != nullptr) {
        free(NETIO);
        NETIO = nullptr;
    }
}
#else
void DCE::qUpSample() {
    hipError_t error;

    error = hipMalloc(&dUPSBUFFER, sizeof(qEnhancedParam_t));
    if (error != hipSuccess) {
        printf("Error dUPSBUFFER hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&dPARAM, sizeof(qEnhancedParam_t));
    if (error != hipSuccess) {
        printf("Error dPARAM hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Run dUpSample_x
    dim3 dimBlock = {30, 30, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, IMG_WIDTH / dimBlock.y, 1};    
    dUpSample_x<<<dimGrid, dimBlock>>>(dNETIO, dUPSBUFFER);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dUpSample_x %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

    // Run dUpSample_y
    dimBlock = {24, 30, 1};
    dimGrid = {IMG_HIGHT / dimBlock.x, IMG_WIDTH / dimBlock.y, 1};
    dUpSample_y<<<dimGrid, dimBlock>>>(dUPSBUFFER, dPARAM);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dUpSample_y %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

    if (dUPSBUFFER != nullptr) {
        error = hipFree(dUPSBUFFER);
        if (error != hipSuccess) {
            printf("Error dUPSBUFFER hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dUPSBUFFER = nullptr;
    }
}
#endif

void DCE::qEnhance()
{
    hipError_t error;

#ifdef CPU_UPSAMPLE
    // Copy PARAM to GPU
    error = hipMalloc(&dPARAM, sizeof(qEnhancedParam_t));
    if (error != hipSuccess) {
        printf("Error dPARAM hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(dPARAM, PARAM, sizeof(qEnhancedParam_t), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("Error dPARAM hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }  
#endif

    // Malloc dOUTDATA
    error = hipMalloc(&dOUTDATA, sizeof(RGBIOData_t));
    if (error != hipSuccess) {
        printf("Error dOUTDATA hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    dim3 dimBlock {24, 30, 1};
    dim3 dimGrid;

    dimGrid.x = (IMG_HIGHT + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (IMG_WIDTH + dimBlock.y - 1) / dimBlock.y;
    dimGrid.z = (IMG_CHANNEL + dimBlock.z - 1) / dimBlock.z;

    dEnhance<<<dimGrid, dimBlock>>>(dNORM, dPARAM, dOUTDATA);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dEnhance %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

    error = hipMemcpy(OUTDATA, dOUTDATA, sizeof(RGBIOData_t), hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Error dOUTDATA hipMemcpy() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Free dOUTDATA, dPARAM, dNORM
    if (dOUTDATA != nullptr) {
        error = hipFree(dOUTDATA);
        if (error != hipSuccess) {
            printf("Error dOUTDATA hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dOUTDATA = nullptr;
    }

    if (dPARAM != nullptr) {
        error = hipFree(dPARAM);
        if (error != hipSuccess) {
            printf("Error dPARAM hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dPARAM = nullptr;
    }

    if (dNORM != nullptr) {
        error = hipFree(dNORM);
        if (error != hipSuccess) {
            printf("Error dNORM hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
        dNORM = nullptr;
    }

    // Free PARAM
    if (PARAM != nullptr) {
        free(PARAM);
        PARAM = nullptr;
    }
}
