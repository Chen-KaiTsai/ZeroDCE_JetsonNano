#include "main.cuh"

void DCE_ZeroCopy::initMem_ZeroCopy() {
    hipHostAlloc((void**)&INDATA, sizeof(RGBIOData_t), hipHostMallocMapped);

    hipHostAlloc((void**)&OUTDATA, sizeof(RGBIOData_t), hipHostMallocMapped);

    hipHostAlloc((void**)&NETIO, sizeof(qNetIO_t), hipHostMallocMapped);
    hipHostAlloc((void**)&UPSBUFFER, sizeof(qEnhancedParam_t), hipHostMallocMapped);
    hipHostAlloc((void**)&PARAM, sizeof(qEnhancedParam_t), hipHostMallocMapped);
    
    hipHostAlloc((void**)&CONVW01, sizeof(qWConv1st_t), hipHostMallocMapped);
    hipHostAlloc((void**)&CONVB01, sizeof(qBConv1st_t), hipHostMallocMapped);
    hipHostAlloc((void**)&CONVW02, sizeof(qWConv2nd_t), hipHostMallocMapped);
    hipHostAlloc((void**)&CONVB02, sizeof(qBConv2nd_t), hipHostMallocMapped);
    hipHostAlloc((void**)&CONVW03, sizeof(qWConv3rd_t), hipHostMallocMapped);
    hipHostAlloc((void**)&CONVB03, sizeof(qBConv3rd_t), hipHostMallocMapped);
}

void DCE_ZeroCopy::cleanMem_ZeroCopy() {
    if (INDATA != nullptr)
        hipFree(INDATA);
    if (OUTDATA != nullptr)
        hipFree(OUTDATA);
    if (NETIO != nullptr)
        hipFree(NETIO);
    if (UPSBUFFER != nullptr)
        hipFree(UPSBUFFER);
    if (PARAM != nullptr)
        hipFree(PARAM);
    
    if (CONVW01 != nullptr)
        hipFree(CONVW01);
    if (CONVB01 != nullptr)
        hipFree(CONVB01);
    if (CONVW02 != nullptr)
        hipFree(CONVW02);
    if (CONVB02 != nullptr)
        hipFree(CONVB02);
    if (CONVW03 != nullptr)
        hipFree(CONVW03);
    if (CONVB03 != nullptr)
        hipFree(CONVB03);
}

void DCE_ZeroCopy::qNormNDownSample_ZeroCopy() {
    hipError_t error;

    error = hipHostGetDevicePointer((void**)&dINDATA,  (void*)INDATA , 0);
    if (error != hipSuccess) {
        printf("Error dINDATA hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&dNORM, sizeof(qNormImg_t));
    if (error != hipSuccess) {
        printf("Error dNORM hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);  
    }

    // Run dNorm kernel
    //constexpr unsigned int jobSize = IMG_HIGHT * IMG_WIDTH * IMG_CHANNEL;

    dim3 dimBlock {24, 40, 1};
    dim3 dimGrid;

    dimGrid.x = (IMG_HIGHT + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (IMG_WIDTH + dimBlock.y - 1) / dimBlock.y;
    dimGrid.z = (IMG_CHANNEL + dimBlock.z - 1) / dimBlock.z;

    //printf("dimBlock {%d, %d, %d} dimGrid {%d, %d, %d}\n\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

    dNorm<<<dimGrid, dimBlock>>>(dINDATA, dNORM);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dNorm %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer((void**)&dNETIO,  (void*)NETIO , 0);
    if (error != hipSuccess) {
        printf("Error dNETIO hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    // Run dDownSample kernel
    dimBlock = {30, 32, 1};
    dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};

    dDownSample<<<dimGrid, dimBlock>>>(dNORM, dNETIO);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dDownSample %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void DCE_ZeroCopy::qConv1st_ZeroCopy() {
    hipError_t error;

    error = hipHostGetDevicePointer((void**)&dCONVW01,  (void*)CONVW01 , 0);
    if (error != hipSuccess) {
        printf("Error dCONVW01 hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer((void**)&dCONVB01,  (void*)CONVB01 , 0);
    if (error != hipSuccess) {
        printf("Error dCONVB01 hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&dFEATURE1, sizeof(qNetFeature_t));
    if (error != hipSuccess) {
        printf("Error dFEATURE1 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Run dConv1st kernel
    dim3 dimBlock = {18, 40, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};

    dConv1st<<<dimGrid, dimBlock>>>(dNETIO, dCONVW01, dCONVB01, dFEATURE1);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dConv1st %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void DCE_ZeroCopy::qConv2nd_ZeroCopy() {
    hipError_t error;

    error = hipHostGetDevicePointer((void**)&dCONVW02,  (void*)CONVW02 , 0);
    if (error != hipSuccess) {
        printf("Error dCONVW02 hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer((void**)&dCONVB02,  (void*)CONVB02 , 0);
    if (error != hipSuccess) {
        printf("Error dCONVB02 hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipMalloc(&dFEATURE2, sizeof(qNetFeature_t));
    if (error != hipSuccess) {
        printf("Error dFEATURE2 hipMalloc() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Run dConv1st kernel
    dim3 dimBlock = {18, 40, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};

    dConv2nd<<<dimGrid, dimBlock>>>(dFEATURE1, dCONVW02, dCONVB02, dFEATURE2);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dConv2nd %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void DCE_ZeroCopy::qConv3rd_ZeroCopy() {
    hipError_t error;

    error = hipHostGetDevicePointer((void**)&dCONVW03,  (void*)CONVW03 , 0);
    if (error != hipSuccess) {
        printf("Error dCONVW03 hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer((void**)&dCONVB03,  (void*)CONVB03 , 0);
    if (error != hipSuccess) {
        printf("Error dCONVB03 hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Run dConv3rd
    dim3 dimBlock = {18, 40, 1};
    dim3 dimGrid = {DCE_HEIGHT / dimBlock.x, DCE_WIDTH / dimBlock.y, 1};
    dConv3rd<<<dimGrid, dimBlock>>>(dFEATURE1, dFEATURE2, dCONVW03, dCONVB03, dNETIO);

    if (dFEATURE1 != nullptr) {
        error = hipFree(dFEATURE1);
        if (error != hipSuccess) {
            printf("Error dFEATURE1 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
    }
    if (dFEATURE2 != nullptr) {
        error = hipFree(dFEATURE2);
        if (error != hipSuccess) {
            printf("Error dFEATURE2 hipFree() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
            exit(EXIT_FAILURE);  
        }
    }
}

void DCE_ZeroCopy::qUpSample_ZeroCopy() {
    int coef[12] = {42, 128, 213, 298, 384, 469, 554, 640, 725, 810, 896, 981};
	for(int h = 0; h < DCE_HEIGHT; ++h)
	{
		int wi = 0;
		for(int d = 0; d < DSRATE / 2; ++d, ++wi)
		{
			UPSBUFFER->data[h][wi][0] = NETIO->data[h][0][0];
			UPSBUFFER->data[h][wi][1] = NETIO->data[h][0][1];
			UPSBUFFER->data[h][wi][2] = NETIO->data[h][0][2];
		}
		for(int w = 1; w < DCE_WIDTH; ++w)
		{
			for(int d = 0; d < DSRATE; ++d, ++wi)
			{
				UPSBUFFER->data[h][wi][0] = (coef[d] * (NETIO->data[h][w][0] - NETIO->data[h][w - 1][0]) >> 10) + NETIO->data[h][w - 1][0];
				UPSBUFFER->data[h][wi][1] = (coef[d] * (NETIO->data[h][w][1] - NETIO->data[h][w - 1][1]) >> 10) + NETIO->data[h][w - 1][1];
				UPSBUFFER->data[h][wi][2] = (coef[d] * (NETIO->data[h][w][2] - NETIO->data[h][w - 1][2]) >> 10) + NETIO->data[h][w - 1][2];
			}
		}
		for(int d = 0; d < DSRATE / 2; ++d, ++wi)
		{
			UPSBUFFER->data[h][wi][0] = NETIO->data[h][DCE_WIDTH - 1][0];
			UPSBUFFER->data[h][wi][1] = NETIO->data[h][DCE_WIDTH - 1][1];
			UPSBUFFER->data[h][wi][2] = NETIO->data[h][DCE_WIDTH - 1][2];
		}
	}

	int hi = 0;
	for(int d = 0; d < DSRATE / 2; ++d, ++hi)
	{
		for(int w = 0; w < IMG_WIDTH; ++w)
		{
			PARAM->data[hi][w][0] = UPSBUFFER->data[0][w][0];
			PARAM->data[hi][w][1] = UPSBUFFER->data[0][w][1];
			PARAM->data[hi][w][2] = UPSBUFFER->data[0][w][2];
		}
	}
	for(int h = 1; h < DCE_HEIGHT; ++h)
	{
		for(int d = 0; d < DSRATE; ++d, ++hi)
		{
			for(int w = 0; w < IMG_WIDTH; ++w)
			{
				PARAM->data[hi][w][0] = (coef[d] * (UPSBUFFER->data[h][w][0] - UPSBUFFER->data[h - 1][w][0]) >> 10) + UPSBUFFER->data[h - 1][w][0];
				PARAM->data[hi][w][1] = (coef[d] * (UPSBUFFER->data[h][w][1] - UPSBUFFER->data[h - 1][w][1]) >> 10) + UPSBUFFER->data[h - 1][w][1];
				PARAM->data[hi][w][2] = (coef[d] * (UPSBUFFER->data[h][w][2] - UPSBUFFER->data[h - 1][w][2]) >> 10) + UPSBUFFER->data[h - 1][w][2];
			}

		}
	}
	for(int d = 0; d < DSRATE / 2; ++d, ++hi)
	{
		for(int w = 0; w < IMG_WIDTH; ++w)
		{
			PARAM->data[hi][w][0] = UPSBUFFER->data[DCE_HEIGHT - 1][w][0];
			PARAM->data[hi][w][1] = UPSBUFFER->data[DCE_HEIGHT - 1][w][1];
			PARAM->data[hi][w][2] = UPSBUFFER->data[DCE_HEIGHT - 1][w][2];
		}
	}
}

void DCE_ZeroCopy::qEnhance_ZeroCopy()
{
    hipError_t error;

    error = hipHostGetDevicePointer((void**)&dPARAM,  (void*)PARAM , 0);
    if (error != hipSuccess) {
        printf("Error dPARAM hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error = hipHostGetDevicePointer((void**)&dOUTDATA,  (void*)OUTDATA , 0);
    if (error != hipSuccess) {
        printf("Error dOUTDATA hipHostGetDevicePointer() : %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    dim3 dimBlock {24, 30, 1};
    dim3 dimGrid;

    dimGrid.x = (IMG_HIGHT + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (IMG_WIDTH + dimBlock.y - 1) / dimBlock.y;
    dimGrid.z = (IMG_CHANNEL + dimBlock.z - 1) / dimBlock.z;

    dEnhance<<<dimGrid, dimBlock>>>(dNORM, dPARAM, dOUTDATA);

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("Error: dEnhance %d\n%s\n\n", static_cast<int>(error), hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}
